#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>

// CUDA kernel: simple rolling mean with window=2 (toy example)
__global__ void rolling_mean_kernel(const float* __restrict__ x, float* __restrict__ y, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N - 1) {
        y[i] = (x[i] + x[i+1]) / 2.0f;
    }
}

// C++ wrapper
torch::Tensor rolling_mean(torch::Tensor x) {
    auto y = torch::zeros_like(x);
    int N = x.size(0);
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    rolling_mean_kernel<<<blocks, threads>>>(x.data_ptr<float>(), y.data_ptr<float>(), N);
    return y;
}

// Bindings
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("rolling_mean", &rolling_mean, "Rolling mean (CUDA)");
}
