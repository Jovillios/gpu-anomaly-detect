#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>

// CUDA kernel: simple rolling mean with window=2 (toy example)
__global__ void rolling_mean_kernel(const float* __restrict__ x, float* __restrict__ y, int N, int W) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return ;

    int start = max(0, idx - W + 1);
    int count = idx - start + 1;
    float sum = 0.0;

    for (int j=start; j <= idx; j++)
    {
        sum += x[j];
    }
    y[idx] = sum / count;
}

// C++ wrapper
torch::Tensor rolling_mean(torch::Tensor x, int W) {
    auto y = torch::zeros_like(x);
    int N = x.size(0);
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    rolling_mean_kernel<<<blocks, threads>>>(x.data_ptr<float>(), y.data_ptr<float>(), N, W);
    return y;
}

// Bindings
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("rolling_mean", &rolling_mean, "Rolling mean (CUDA)");
}
